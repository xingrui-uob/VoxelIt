#include "hip/hip_runtime.h"
#include "MapStruct.h"
#include "MapStructFuncs.h"
#include <opencv2/opencv.hpp>

namespace voxelization
{

__global__ void resetHashKernel(HashEntry *hashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    hashTable[index].ptr = -1;
    hashTable[index].offset = -1;
}

__global__ void resetHeapKernel(int *heap, int *heapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        heapPtr[0] = numBlock - 1;

    heap[index] = numBlock - index - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(cv::divUp(nEntry, block.x));
    resetHashKernel<<<grid, block>>>(hashTable, nEntry);

    grid = dim3(cv::divUp(nVBlock, block.x));
    resetHeapKernel<<<grid, block>>>(heap, heapPtr, nVBlock);

    hipMemset(excessPtr, 0, sizeof(int));
    hipMemset(bucketMutex, 0, sizeof(int) * nBucket);
    hipMemset(voxelBlock, 0, sizeof(Voxel) * BLOCK_SIZE_3 * nVBlock);
}

void MapStruct::create(
    int nEntry,
    int nBucket,
    int nVBlock,
    float voxelSize,
    float truncationDist)
{
    hipMalloc((void **)&excessPtr, sizeof(int));
    hipMalloc((void **)&heapPtr, sizeof(int));
    hipMalloc((void **)&visibleBlockNum, sizeof(uint));
    hipMalloc((void **)&bucketMutex, sizeof(int) * nBucket);
    hipMalloc((void **)&heap, sizeof(int) * nVBlock);
    hipMalloc((void **)&hashTable, sizeof(HashEntry) * nEntry);
    hipMalloc((void **)&visibleTable, sizeof(HashEntry) * nEntry);
    hipMalloc((void **)&voxelBlock, sizeof(Voxel) * nVBlock * BLOCK_SIZE_3);

    this->nEntry = nEntry;
    this->nBucket = nBucket;
    this->nVBlock = nVBlock;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

void MapStruct::release()
{
    hipFree((void *)heap);
    hipFree((void *)heapPtr);
    hipFree((void *)hashTable);
    hipFree((void *)bucketMutex);
    hipFree((void *)excessPtr);
    hipFree((void *)voxelBlock);
    hipFree((void *)visibleBlockNum);
    hipFree((void *)visibleTable);
}

void MapStruct::getVisibleBlockCount(uint &hostData)
{
    hipMemcpy(&hostData, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost);
}

void MapStruct::resetVisibleBlockCount()
{
    hipMemset(visibleBlockNum, 0, sizeof(uint));
}

bool MapStruct::empty()
{
    return nBucket == 0;
}

} // namespace voxelization