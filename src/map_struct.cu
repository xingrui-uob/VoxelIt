#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "map_struct.h"
#include <opencv2/opencv.hpp>

namespace vmap
{
__global__ void resetHashKernel(HashEntry* hashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    hashTable[index].ptr = -1;
    hashTable[index].offset = -1;
}

__global__ void resetHeapKernel(int* heap, int* heapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        heapPtr[0] = numBlock - 1;

    heap[index] = numBlock - index - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(cv::divUp(nEntry, block.x));
    resetHashKernel<<<grid, block>>>(hashTable, nEntry);

    grid = dim3(cv::divUp(nVBlock, block.x));
    resetHeapKernel<<<grid, block>>>(heap, heapPtr, nVBlock);

    SafeCall(hipMemset(excessPtr, 0, sizeof(int)));
    SafeCall(hipMemset(bucketMutex, 0, sizeof(int) * nBucket));
    SafeCall(hipMemset(voxelBlock, 0, sizeof(Voxel) * BLOCK_SIZE_3 * nVBlock));
}

void MapStruct::create(
    int nEntry,
    int nBucket,
    int nVBlock,
    float voxelSize,
    float truncationDist)
{
    SafeCall(hipMalloc((void**)&excessPtr, sizeof(int)));
    SafeCall(hipMalloc((void**)&heapPtr, sizeof(int)));
    SafeCall(hipMalloc((void**)&visibleBlockNum, sizeof(uint)));
    SafeCall(hipMalloc((void**)&bucketMutex, sizeof(int) * nBucket));
    SafeCall(hipMalloc((void**)&heap, sizeof(int) * nVBlock));
    SafeCall(hipMalloc((void**)&hashTable, sizeof(HashEntry) * nEntry));
    SafeCall(hipMalloc((void**)&visibleTable, sizeof(HashEntry) * nEntry));
    SafeCall(hipMalloc((void**)&voxelBlock, sizeof(Voxel) * nVBlock * BLOCK_SIZE_3));

    this->nEntry = nEntry;
    this->nBucket = nBucket;
    this->nVBlock = nVBlock;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

void MapStruct::release()
{
    SafeCall(hipFree((void*)heap));
    SafeCall(hipFree((void*)heapPtr));
    SafeCall(hipFree((void*)hashTable));
    SafeCall(hipFree((void*)bucketMutex));
    SafeCall(hipFree((void*)excessPtr));
    SafeCall(hipFree((void*)voxelBlock));
    SafeCall(hipFree((void*)visibleBlockNum));
    SafeCall(hipFree((void*)visibleTable));
}

void MapStruct::getVisibleBlockCount(uint& hostData)
{
    SafeCall(hipMemcpy(&hostData, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost));
}

void MapStruct::resetVisibleBlockCount()
{
    SafeCall(hipMemset(visibleBlockNum, 0, sizeof(uint)));
}

bool MapStruct::empty()
{
    return nBucket == 0;
}

} // namespace vmap